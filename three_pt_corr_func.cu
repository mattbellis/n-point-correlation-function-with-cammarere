#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<math.h>


// The user may want to change this 
// This defines the number of bins in the histogram
#define imax 128
//#define imax 16
#define FLOAT_RAND_MAX (float)RAND_MAX

/////////////////////////////////////////////////////////////
void binning3d_CPU(float *x0, float *y0, float *z0,  int num_vals_to_bin, float lo, float hi, int nbins, float binwidth, int *bin_indices) {

	int bin = -1;
	float v;

	// The user has to make sure that filled_bins has enough memory allocated
	// for nbins integers

	for (int idx = 0; idx<num_vals_to_bin; idx++ ) {

		v = vals[idx];
		//v = sin(log(pow(1000,v)));
		//v = sin(log(pow(1000,v)));
		//v = sin(log(pow(1000,v)));
		//v = sin(log(pow(1000,v)));
        //v = abs(v);

		// DEBUG PRINT
        //printf("%f\n",v);

		bin = -1;
		if (v<lo) {
			bin=-1;
		}
		else if (v>hi) {
			bin = -999;
		}
		else {
			bin = (int)((v-lo)/binwidth);
		}   
		//bin = idx; // for DEBUGGING
		bin_indices[idx] = bin;
	}

	// bin_indices is a pointer so the values in it will 
	// still be accessible outside of the function.

}

__global__ void binning3d(float *vals, int num_vals_to_bin, float lo, float hi, int nbins, float binwidth, int *bin_indices) {

	int bin = -1;
	float v;

	// The user has to make sure that filled_bins has enough memory allocated
	// for nbins integers

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx<num_vals_to_bin) {

		v = vals[idx];
		//v = sin(log(pow(1000,v)));
		//v = sin(log(pow(1000,v)));
		//v = sin(log(pow(1000,v)));
		//v = sin(log(pow(1000,v)));
        //v = abs(v);
		bin = -1;
		if (v<lo) {
			bin=-1;
		}
		else if (v>hi) {
			bin = -999;
		}
		else {
			bin = (int)((v-lo)/binwidth);
		}   
		//bin = idx; // for DEBUGGING
		bin_indices[idx] = bin;
	}

	// bin_indices is a pointer so the values in it will 
	// still be accessible outside of the function.

}

///////////////////////////////////////////////////////////////////////////////
int main(int argc, char *argv[]) {

	// How many random values do we want to process? 
	// This can be set on the command line or you can just
	// edit it and recompile each time
	bool GPU_FLAG = false;
	char* CPU_OR_GPU = argv[1];
	if (strcmp(CPU_OR_GPU,"gpu")==0) {
		GPU_FLAG = true;
	}
	else if (strcmp(CPU_OR_GPU,"cpu")!=0 && strcmp(CPU_OR_GPU,"gpu")!=0) {
		printf("First argument must be 'gpu' or 'cpu'!\n");
		exit(-1);
	}

	unsigned long nvals = atoll(argv[2]);
	printf("nvals: %lu\n",nvals);


	// For the random numbers, they will be between 0 and 1
	float lo = 0;
	float hi = 1;
	int nbins = imax;
	float binwidth = (hi-lo)/nbins;
	// This means we'll send this many values to the function
	// to be histogrammed
	//int histogram_chunks = 16*16; 
	//int histogram_chunks = 256*256; 
	int histogram_chunks = 512*512; 
	//int histogram_chunks = 1024*1024; 

	printf("Filling a histogram with\n");
	printf("Range: %f-%f\n",lo,hi);
	printf("# of bins: %d\n",nbins);
	printf("Bin width: %f\n",binwidth);
	printf("We will histogram in chunks of: %d\n",histogram_chunks);

	// These will be the values we're histogramming
	// I think this is wrong and tries to allocate too much memory
	float *x0 = (float*) malloc(histogram_chunks * sizeof(float));
	float *y0 = (float*) malloc(histogram_chunks * sizeof(float));
	float *z0 = (float*) malloc(histogram_chunks * sizeof(float));
	float *x1 = (float*) malloc(histogram_chunks * sizeof(float));
	float *y1 = (float*) malloc(histogram_chunks * sizeof(float));
	float *z1 = (float*) malloc(histogram_chunks * sizeof(float));

	int *bin_indices = (int*) malloc(histogram_chunks * sizeof(int));

	float *d_x0, *d_y0, *d_z0, *d_x1, *d_y1, *d_z1;
	int *d_bin_indices;
	hipMalloc ((void**)&d_x0, histogram_chunks * sizeof(float));
	hipMalloc ((void**)&d_y0, histogram_chunks * sizeof(float));
	hipMalloc ((void**)&d_z0, histogram_chunks * sizeof(float));
	hipMalloc ((void**)&d_x1, histogram_chunks * sizeof(float));
	hipMalloc ((void**)&d_y1, histogram_chunks * sizeof(float));
	hipMalloc ((void**)&d_z1, histogram_chunks * sizeof(float));
	hipMalloc ((void**)&d_bin_indices, histogram_chunks * sizeof(int));

	hipMemset (d_bin_indices, -1, histogram_chunks * sizeof(int));

	int i = 0; 

	static unsigned long hist[imax];

	printf("Allocated the memory for the histogram.\n");

	// Zero out the entries in the histogram
	for (i=0;i<imax;i++) {
		hist[i] = 0;
	}

	printf("Zeroed the memory in the histogram.\n");

	printf("Filling the memory with %lu entries .\n",nvals);
	printf("RAND_MAX: %d\n",RAND_MAX);

	int count_for_histogramming = 0;

	int ichunk = 0;
	for(unsigned long count=0;count<nvals;count++)
	{
		//if (count%10000000==0){
			//printf("count: %lu\n",count);
		//}

		// Fill the array of values that we will histogram
		//if (ichunk==0) 
			x0[count_for_histogramming] = rand()/FLOAT_RAND_MAX;
			y0[count_for_histogramming] = rand()/FLOAT_RAND_MAX;
			z0[count_for_histogramming] = rand()/FLOAT_RAND_MAX;
			x1[count_for_histogramming] = rand()/FLOAT_RAND_MAX;
			y1[count_for_histogramming] = rand()/FLOAT_RAND_MAX;
			z1[count_for_histogramming] = rand()/FLOAT_RAND_MAX;
		// A lot of time is spent generating random numbers.

		// DEBUG PRINT
		//printf("This point: %lu %f\n",count,x0[count_for_histogramming]);

		// Keep track of this by hand
		count_for_histogramming++;

		// When we have enough, go histogram them!
		if (count_for_histogramming==histogram_chunks || count==nvals-1) {
			//printf("Histogramming a chunk of values!\n");
			//printf("histogram_chunks: %d\n",histogram_chunks);
			//printf("count_for_histogramming: %d\n",count_for_histogramming);

			if (GPU_FLAG) {
				// Copy over
				//printf("Using the GPU!\n");
				//if (ichunk==0) {
					hipMemcpy(d_x0, x0, sizeof(float) * histogram_chunks, hipMemcpyHostToDevice);
					hipMemcpy(d_y0, y0, sizeof(float) * histogram_chunks, hipMemcpyHostToDevice);
					hipMemcpy(d_z0, z0, sizeof(float) * histogram_chunks, hipMemcpyHostToDevice);
					hipMemcpy(d_x1, x1, sizeof(float) * histogram_chunks, hipMemcpyHostToDevice);
					hipMemcpy(d_y1, y1, sizeof(float) * histogram_chunks, hipMemcpyHostToDevice);
					hipMemcpy(d_z1, z1, sizeof(float) * histogram_chunks, hipMemcpyHostToDevice);
				//	printf("Copied!\n");
				//}

				binning3d<<<1024,1024>>>(d_x0, count_for_histogramming, lo, hi, nbins, binwidth, d_bin_indices);

				hipMemcpy(bin_indices, d_bin_indices, sizeof(int) * histogram_chunks, hipMemcpyDeviceToHost);
			}
			else {
				binning3d_CPU(x0, count_for_histogramming, lo, hi, nbins, binwidth, bin_indices);
			}

			for (int j=0;j<count_for_histogramming;j++) {
				// DEBUG PRINT
				//printf("Summing: %d %d\n",j,bin_indices[j]);
				if (bin_indices[j]>=0 && bin_indices[j]<nbins)
					hist[bin_indices[j]]++;
			}
			// Reset the counter
			count_for_histogramming = 0;
			ichunk++;
		}
	}

	// Print out the histogram
	printf("Printing out the histogram entries\n");
	unsigned long total = 0;
	for (i = 0; i <  imax; i++)  {
        // DEBUG PRINT
		//printf("%d %lu\n",i, hist[i]);
		total += hist[i];
	}
	printf("Total entries: %lu\n",total);

	hipFree(d_x0);
	hipFree(d_bin_indices);
	free(x0);
	free(bin_indices);

	return 0;
}
