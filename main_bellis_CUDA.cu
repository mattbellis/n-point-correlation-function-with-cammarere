
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>


// The user may want to change this 
// This defines the number of bins in the histogram
#define imax 128
//#define imax 16
#define FLOAT_RAND_MAX (float)RAND_MAX

/////////////////////////////////////////////////////////////
void binning1d_CPU(float *vals, int num_vals_to_bin, float lo, float hi, int nbins, float binwidth, int *bin_indices) {

	int bin = -1;
	float v;

	// The user has to make sure that filled_bins has enough memory allocated
	// for nbins integers

	for (int idx = 0; idx<num_vals_to_bin; idx++ ) {

		v = vals[idx];
		//v = sin(log(pow(1000,v)));
		//v = sin(log(pow(1000,v)));
		//v = sin(log(pow(1000,v)));
		//v = sin(log(pow(1000,v)));
        //v = abs(v);

		// DEBUG PRINT
        //printf("%f\n",v);

		bin = -1;
		if (v<lo) {
			bin=-1;
		}
		else if (v>hi) {
			bin = -999;
		}
		else {
			bin = (int)((v-lo)/binwidth);
		}   
		//bin = idx; // for DEBUGGING
		bin_indices[idx] = bin;
	}

	// bin_indices is a pointer so the values in it will 
	// still be accessible outside of the function.

}

__global__ void binning1d(float *vals, int num_vals_to_bin, float lo, float hi, int nbins, float binwidth, int *bin_indices) {

	int bin = -1;
	float v;

	// The user has to make sure that filled_bins has enough memory allocated
	// for nbins integers

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx<num_vals_to_bin) {

		v = vals[idx];
		//v = sin(log(pow(1000,v)));
		//v = sin(log(pow(1000,v)));
		//v = sin(log(pow(1000,v)));
		//v = sin(log(pow(1000,v)));
        //v = abs(v);
		bin = -1;
		if (v<lo) {
			bin=-1;
		}
		else if (v>hi) {
			bin = -999;
		}
		else {
			bin = (int)((v-lo)/binwidth);
		}   
		//bin = idx; // for DEBUGGING
		bin_indices[idx] = bin;
	}

	// bin_indices is a pointer so the values in it will 
	// still be accessible outside of the function.

}

///////////////////////////////////////////////////////////////////////////////
int main(int argc, char *argv[]) {

	// How many random values do we want to process? 
	// This can be set on the command line or you can just
	// edit it and recompile each time
	bool GPU_FLAG = false;
	char* CPU_OR_GPU = argv[1];
	if (strcmp(CPU_OR_GPU,"gpu")==0) {
		GPU_FLAG = true;
	}
	else if (strcmp(CPU_OR_GPU,"cpu")!=0 && strcmp(CPU_OR_GPU,"gpu")!=0) {
		printf("First argument must be 'gpu' or 'cpu'!\n");
		exit(-1);
	}

	unsigned long nvals = atoll(argv[2]);
	printf("nvals: %lu\n",nvals);


	// For the random numbers, they will be between 0 and 1
	float lo = 0;
	float hi = 1;
	int nbins = imax;
	float binwidth = (hi-lo)/nbins;
	// This means we'll send this many values to the function
	// to be histogrammed
	//int histogram_chunks = 16*16; 
	//int histogram_chunks = 256*256; 
	int histogram_chunks = 512*512; 
	//int histogram_chunks = 1024*1024; 

	printf("Filling a histogram with\n");
	printf("Range: %f-%f\n",lo,hi);
	printf("# of bins: %d\n",nbins);
	printf("Bin width: %f\n",binwidth);
	printf("We will histogram in chunks of: %d\n",histogram_chunks);

	// These will be the values we're histogramming
	// I think this is wrong and tries to allocate too much memory
	//float *values_to_be_histogrammed = (float*) malloc(nvals * sizeof(float));
	//unsigned long *bin_indices = (unsigned long*) malloc(nvals * sizeof(unsigned long));
	float *values_to_be_histogrammed = (float*) malloc(histogram_chunks * sizeof(float));
	int *bin_indices = (int*) malloc(histogram_chunks * sizeof(int));

	float *d_values_to_be_histogrammed;
	int *d_bin_indices;
	hipMalloc ((void**)&d_values_to_be_histogrammed, histogram_chunks * sizeof(float));
	hipMalloc ((void**)&d_bin_indices, histogram_chunks * sizeof(int));

	hipMemset (d_bin_indices, -1, histogram_chunks * sizeof(int));

	int i = 0; 

	static unsigned long hist[imax];

	printf("Allocated the memory for the histogram.\n");

	// Zero out the entries in the histogram
	for (i=0;i<imax;i++) {
		hist[i] = 0;
	}

	printf("Zeroed the memory in the histogram.\n");

	printf("Filling the memory with %lu entries .\n",nvals);
	printf("RAND_MAX: %d\n",RAND_MAX);

	int count_for_histogramming = 0;

	int ichunk = 0;
	for(unsigned long count=0;count<nvals;count++)
	{
		//if (count%10000000==0){
			//printf("count: %lu\n",count);
		//}

		// Fill the array of values that we will histogram
		//if (ichunk==0) 
			values_to_be_histogrammed[count_for_histogramming] = rand()/FLOAT_RAND_MAX;
		// A lot of time is spent generating random numbers.

		// DEBUG PRINT
		//printf("This point: %lu %f\n",count,values_to_be_histogrammed[count_for_histogramming]);

		// Keep track of this by hand
		count_for_histogramming++;

		// When we have enough, go histogram them!
		if (count_for_histogramming==histogram_chunks || count==nvals-1) {
			//printf("Histogramming a chunk of values!\n");
			//printf("histogram_chunks: %d\n",histogram_chunks);
			//printf("count_for_histogramming: %d\n",count_for_histogramming);

			if (GPU_FLAG) {
				// Copy over
				//printf("Using the GPU!\n");
				//if (ichunk==0) {
					hipMemcpy(d_values_to_be_histogrammed, values_to_be_histogrammed, sizeof(float) * histogram_chunks, hipMemcpyHostToDevice);
				//	printf("Copied!\n");
				//}

				binning1d<<<1024,1024>>>(d_values_to_be_histogrammed, count_for_histogramming, lo, hi, nbins, binwidth, d_bin_indices);

				hipMemcpy(bin_indices, d_bin_indices, sizeof(int) * histogram_chunks, hipMemcpyDeviceToHost);
			}
			else {
				binning1d_CPU(values_to_be_histogrammed, count_for_histogramming, lo, hi, nbins, binwidth, bin_indices);
			}

			for (int j=0;j<count_for_histogramming;j++) {
				// DEBUG PRINT
				//printf("Summing: %d %d\n",j,bin_indices[j]);
				if (bin_indices[j]>=0 && bin_indices[j]<nbins)
					hist[bin_indices[j]]++;
			}
			// Reset the counter
			count_for_histogramming = 0;
			ichunk++;
		}
	}

	// Print out the histogram
	printf("Printing out the histogram entries\n");
	unsigned long total = 0;
	for (i = 0; i <  imax; i++)  {
        // DEBUG PRINT
		//printf("%d %lu\n",i, hist[i]);
		total += hist[i];
	}
	printf("Total entries: %lu\n",total);

	hipFree(d_values_to_be_histogrammed);
	hipFree(d_bin_indices);
	free(values_to_be_histogrammed);
	free(bin_indices);

	return 0;
}
